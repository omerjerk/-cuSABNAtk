#include "hip/hip_runtime.h"
#include "gpu_util.cuh"
#include <cstdio>

//
// reduction based on cuda-8.0/samples/6_Advanced/reduction
//

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
counts(const T **g_idata,
  T *g_odata,
  T *g_rdata,
  unsigned int words_per_vector,
  const int vectors_per_config,
  const int configs_per_query)
{
  T *sdata = SharedMemory<T>();

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockSize + threadIdx.x;
  unsigned int vector_index = vectors_per_config * blockIdx.x;
  unsigned int word_index = i%blockSize; // cant this be tid
  unsigned int result_index = blockIdx.x * words_per_vector + tid;

  T mySum = 0;
  T localState = g_idata[vector_index][word_index]; // first word slice of config

  //printf("%d %d %d %d %d\n", result_index, tid, blockIdx.x, blockSize, words_per_vector);

  // running sum for all word slices
  for(int p = 1; p < vectors_per_config; p++)
  {
    localState = localState & g_idata[vector_index + p][word_index];
  }

  if(g_rdata != 0){ // todo can be compile time decision
    g_rdata[result_index] = localState;
  }

  mySum += __popcll(localState);

  // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
  if (nIsPow2 || (tid + blockSize < words_per_vector) )
  {
    unsigned int word_index_upper_half = word_index + blockSize;
    localState = g_idata[vector_index][word_index_upper_half];

    for(int p = 1; p < vectors_per_config; p++)
    {
      localState = localState & g_idata[vector_index + p][word_index_upper_half];
    }

    if(g_rdata != 0){ // todo can be compile time decision
      g_rdata[result_index + blockSize] = localState;
    }

//    printf("%d %d %d %d %d\n", result_index + blockSize, tid, blockIdx.x, blockSize, words_per_vector);

    mySum += __popcll(localState);
  }

   // each thread puts its local sum into shared memory
  sdata[tid] = mySum;

  __syncthreads();

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256))
  {
      sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  __syncthreads();

  if ((blockSize >= 256) &&(tid < 128))
  {
      sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

   __syncthreads();

  if ((blockSize >= 128) && (tid <  64))
  {
     sdata[tid] = mySum = mySum + sdata[tid +  64];
  }

  __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
  if ( tid < 32 )
  {
      // Fetch final intermediate sum from 2nd warp
      if (blockSize >=  64) mySum += sdata[tid + 32];
      // Reduce final warp using shuffle
      for (int offset = warpSize/2; offset > 0; offset /= 2)
      {
          mySum += __shfl_down(mySum, offset);
      }
  }
#else
  // fully unroll reduction within a single warp
  if ((blockSize >=  64) && (tid < 32))
  {
      sdata[tid] = mySum = mySum + sdata[tid + 32];
  }

  __syncthreads();

  if ((blockSize >=  32) && (tid < 16))
  {
      sdata[tid] = mySum = mySum + sdata[tid + 16];
  }

  __syncthreads();

  if ((blockSize >=  16) && (tid <  8))
  {
      sdata[tid] = mySum = mySum + sdata[tid +  8];
  }

  __syncthreads();

  if ((blockSize >=   8) && (tid <  4))
  {
      sdata[tid] = mySum = mySum + sdata[tid +  4];
  }

  __syncthreads();

  if ((blockSize >=   4) && (tid <  2))
  {
      sdata[tid] = mySum = mySum + sdata[tid +  2];
  }

  __syncthreads();

  if ((blockSize >=   2) && ( tid <  1))
  {
      sdata[tid] = mySum = mySum + sdata[tid +  1];
  }

  __syncthreads();
#endif

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = mySum;
  __syncthreads();
} // counts

// from cuda samples reduction
unsigned int nextPow2(unsigned int x);

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}
bool isPow2(unsigned int x);

bool isPow2(unsigned int x)
{
  return ((x&(x-1))==0);
}

void cudaCallBlockCount(
  const uint block_count,
  const uint per_block_thread_count,
  const uint words_per_vector,
  const uint vectors_per_config,
  const uint configs_per_query,
  const unsigned long long** bvectorsPtr,
  unsigned long long* results,
  unsigned long long* states) {

    int threads = nextPow2((words_per_vector + 1)/ 2);

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(configs_per_query, 1, 1);
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(unsigned long long) : threads * sizeof(unsigned long long);

    if (isPow2(words_per_vector) && (words_per_vector > 1)) // optimize out non pwr of 2 logic
    {
      switch (threads)
      {
        case 512:
        counts<unsigned long long, 512, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 256:
        counts<unsigned long long, 256, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 128:
        counts<unsigned long long, 128, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 64:
        counts<unsigned long long, 64, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 32:
        counts<unsigned long long, 32, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 16:
        counts<unsigned long long, 16, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 8:
        counts<unsigned long long, 8, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 4:
        counts<unsigned long long, 4, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 2:
        counts<unsigned long long, 2, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 1:
        counts<unsigned long long, 1, true><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;
      }
    }
    else
    {
      switch (threads)
      {
        case 512:
        counts<unsigned long long, 512, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 256:
        counts<unsigned long long, 256, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 128:
        counts<unsigned long long, 128, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 64:
        counts<unsigned long long, 64, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 32:
        counts<unsigned long long, 32, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 16:
        counts<unsigned long long, 16, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 8:
        counts<unsigned long long, 8, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 4:
        counts<unsigned long long, 4, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 2:
        counts<unsigned long long, 2, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;

        case 1:
        counts<unsigned long long, 1, false><<< dimGrid, dimBlock, smemSize >>>(bvectorsPtr, results, states, words_per_vector, vectors_per_config, configs_per_query); //words, bvs/config, configs per query
        break;
      }
    }
  }
